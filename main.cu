#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#include "volume.h"
#include "exhaustiveFT.h"
#include "maurer.h"
#include "maurer_GPU.cuh"

/*
	Testing Functions are Defined here
*/
void initVolume(uchar *, int, int, int);
void printVolume(uchar *, int, int, int);
int check(int *, int *, int);

void checkResult(uchar *input, int *vol, int height, int width, int depth);
double calcDist(double i, double j, double target_i, double target_j);

bool isKthBitSet(int n, int k) 
{ 
    if (n & (1 << (k - 1))) 
        return 1; 
    else
        return 0; 
}

int main()
{
	// Initialize Input and Output Data
	/*	ref will always stay in host */
	int *ref = (int *)malloc(HEIGHT * WIDTH * DEPTH * sizeof(int));
	if (ref == NULL)
	{
		printf("malloc() failed! \n");
		return 0;
	}

	/* Input will be transfered to GPU */
	uchar *input;
	if (hipHostAlloc((void **)&input, HEIGHT * WIDTH * DEPTH * sizeof(uchar), \
						hipHostMallocDefault) != hipSuccess)
	{
		printf("hipHostAlloc() failed! \n");
		return 0;
	}

	/* Output will be transfered back to CPU */
	int *output_0;
	if (hipHostAlloc((void **)&output_0, HEIGHT * WIDTH * DEPTH * sizeof(int), \
						hipHostMallocDefault) != hipSuccess)
	{
		printf("hipHostAlloc() failed! \n");
		return 0;
	}

	int *output_1;
	if (hipHostAlloc((void **)&output_1, HEIGHT * WIDTH * DEPTH * sizeof(int), \
						hipHostMallocDefault) != hipSuccess)
	{
		printf("hipHostAlloc() failed! \n");
		return 0;
	}

	for (int i = 0; i < HEIGHT * WIDTH * DEPTH; i++)
	{
		output_0[i] = -1;
		output_1[i] = -1;
		ref[i] = -1;
	}

	// Generate Testing Data
	initVolume(input, HEIGHT, WIDTH, DEPTH);	

	// Print Volume
	printVolume(input, HEIGHT, WIDTH, DEPTH);

	// Generate Reference Result
	exhaustiveFT(input, HEIGHT, WIDTH, DEPTH, ref);

	

	// Compute using Maurer
	struct timeval stopCPU, startCPU;
	gettimeofday(&startCPU, NULL);
	maurerFT(input, HEIGHT, WIDTH, DEPTH, output_0);
	gettimeofday(&stopCPU, NULL);
	long seconds = stopCPU.tv_sec - startCPU.tv_sec;
	long useconds = stopCPU.tv_usec - startCPU.tv_usec;
	long mtime = seconds * 1000 + useconds / 1000.0;
	//printf("CPU Execution Time: %ld ms. \n", mtime);

	// Check the closet boundary point
		

	checkResult(input, output_0, HEIGHT, WIDTH, DEPTH);

	/*
		GPU Solutions
	*/
	/* CUDA Performance */
	/*
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// Allocate device memory for input data
	uchar *dev_vol;
	if (hipMalloc((void **)&dev_vol, HEIGHT * WIDTH * DEPTH * sizeof(uchar)\
		) != hipSuccess)
	{
		printf("hipMalloc() failed.\n");
		return 0;
	}
	
	// Allocate device memory for ping-pong I/O
	int *dev_ping_pong_buf[2];

	for (int i = 0; i < 2; i++)
	{
		if (hipMalloc((void **)&(dev_ping_pong_buf[i]), HEIGHT * WIDTH * DEPTH * sizeof(int)\
			) != hipSuccess)
		{
			printf("hipMalloc() failed! \n");
			return 0;
		}	
	}

	// Bind texture memory - Remember to Unbind!
	// if (hipBindTexture( NULL, ping_pong_tex_0, dev_ping_pong_buf[0], HEIGHT * WIDTH * DEPTH * sizeof(int)) \
		!= hipSuccess)
	// {
	//	printf("hipBindTexture() failed! \n");
	//	return 0;
	// }

	// Copy input data to GPU
	hipMemcpy(dev_vol, input, HEIGHT * WIDTH * DEPTH * sizeof(uchar), \
		hipMemcpyHostToDevice);

	// Computation
	dim3 dimBlock_D1_V1(256, 2, 2);
	dim3 dimGrid_D1_V1(1, 128, 32);

	dim3 dimBlock_D2_V1(2, 256, 2);
	dim3 dimGrid_D2_V1(128, 1, 32);

	//dim3 dimBlock_D3(4, 4, 64);
	//dim3 dimGrid_D3(64, 64, 1);

	dim3 dimBlock_D1_V2(32, 8, 4);
	dim3 dimGrid_D1_V2(1, 4, 2);
	//dim3 dimGrid_D1_V2(1, 32, 16);

	dim3 dimBlock_D2_V2(8, 32, 4);
	dim3 dimGrid_D2_V2(4, 1, 2);
	//dim3 dimGrid_D2_V2(32, 1, 16);

	//dim3 dimBlock_D3_V2(8, 4, 32);
	//dim3 dimGrid_D3_V2(32, 64, 1);
	dim3 dimBlock_D3_V2(8, 4, 8);
	dim3 dimGrid_D3_V2(4, 8, 1);


	hipEventRecord(start);
	//initVol_GPU<<<360, 256>>>(dev_vol, HEIGHT, WIDTH, DEPTH, dev_output);
	//raster_scan_GPU_v1<<<dimGrid_D1_V1, dimBlock_D1_V1>>>(dev_vol, 1, WIDTH, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[0]);
	raster_scan_GPU_v2<<<dimGrid_D1_V2, dimBlock_D1_V2>>>(dev_vol, 0, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[1]);
	raster_scan_GPU_v2<<<dimGrid_D2_V2, dimBlock_D2_V2>>>(dev_ping_pong_buf[1], 1, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[0]);
	raster_scan_GPU_v2<<<dimGrid_D3_V2, dimBlock_D3_V2>>>(dev_ping_pong_buf[0], 2, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[1]);

	//raster_scan_GPU_v2<<<dimGrid_D3_V2, dimBlock_D3_V2>>>(dev_ping_pong_buf[0], 2, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[1]);
	//raster_scan_GPU_v1<<<dimGrid_D2_V1, dimBlock_D2_V1>>>(dev_ping_pong_buf[0], 2, HEIGHT, HEIGHT, WIDTH, DEPTH, \
												dev_ping_pong_buf[1]);
	// Different buffer is needed since the original data may get overwritten
	//raster_scan_GPU_v2<<<dimGrid_D2_V2, dimBlock_D2_V2>>>(dev_ping_pong_buf[0], 1, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[1]);

	// maurer_GPU<<<dimGrid_D3, dimBlock_D3>>>(dev_vol, 3, DEPTH, HEIGHT, WIDTH, DEPTH, dev_output);
	
	// Send result back to host
	//hipMemcpy(output_0, dev_ping_pong_buf[0], HEIGHT * WIDTH * DEPTH * sizeof(int), \
		hipMemcpyDeviceToHost);
	hipMemcpy(output_1, dev_ping_pong_buf[1], HEIGHT * WIDTH * DEPTH * sizeof(int), \
		hipMemcpyDeviceToHost);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Execution Time on GPU: %f ms\n", milliseconds);

	// Free device resource
	// hipUnbindTexture(ping_pong_tex_0);

	hipFree(dev_vol);
	for (int i = 0; i < 2; i++)
	{
		hipFree(dev_ping_pong_buf[i]);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	*/

	// Check result
	if (check(ref, output_0, HEIGHT * WIDTH * DEPTH) == 0)
	{
		printf("Error! \n");
	}
	else
	{
		printf("Successful! \n");
	}

	/*
	if (check(ref, output_1, HEIGHT * WIDTH * DEPTH) == 0)
	{
		printf("GPU V2 Error! \n");
	}
	else
	{
		printf("GPU V2 Successful! \n");
	}
	*/
	
	// De-allocate Memory
	hipHostFree(input);
	hipHostFree(output_0);
	hipHostFree(output_1);
	free(ref);	
}

void initVolume(uchar *vol, int height, int width, int depth)
{
	for (int i = 0; i < HEIGHT * WIDTH * DEPTH; i++)
	{
		 vol[i] = 0;
	}

	srand(time(NULL));

	// Distance between slices
	int slice_stride = height * width;

	/*
	for (int ite = 0; ite < 5; ite++)
	{
		int r_row = rand() % height;
		int r_col = rand() % width;
		int r_dep = rand() % depth;

		vol[r_dep * slice_stride + r_row * width + r_col] = 1;
	}
	*/
	
	int r_row = 3;
	int r_col = 3;
	int r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x03;

	r_row = 3;
	r_col = 4;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x02;

	r_row = 3;
	r_col = 5;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x0A;

	r_row = 4;
	r_col = 3;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x01;

	r_row = 4;
	r_col = 6;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x0A;

	r_row = 5;
	r_col = 3;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x11;

	r_row = 5;
	r_col = 6;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x08;

	r_row = 6;
	r_col = 4;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x11;
	
	r_row = 6;
	r_col = 5;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x10;

	r_row = 6;
	r_col = 6;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x18;
}

void printVolume(uchar *vol, int height, int width, int depth)
{
	// Distance between slices
	int slice_stride = height * width;

	int i, j, k;

	for (k = 0; k < depth; k++)
	{
		printf("Image Slice: %d\n", k);

		for (i = 0; i < height; i++)
		{
			for (j = 0; j < width; j++)
			{
				printf("0x%02x ", vol[k * slice_stride + i * width + j]);
			}
			printf("\n");
		}
		printf("\n");
	}
}

void checkResult(uchar *input, int *vol, int height, int width, int depth)
{
	// Distance between slices
	int slice_stride = height * width;

	int i, j, k;

	for (k = 0; k < depth; k++)
	{
		printf("Image Slice: %d\n", k);

		for (i = 0; i < height; i++)
		{
			for (j = 0; j < width; j++)
			{
				int row_id = vol[k * slice_stride + i * width + j] / width;
				int col_id = vol[k * slice_stride + i * width + j] % width;
				
				if (row_id == i && col_id == j)
				{
					printf("**********  ");
				}
				else
				{
					int input_idx = vol[k * slice_stride + i * width + j];
					//printf("(%d, %d) \n", row_id, col_id);
					//printf("Value: %d \n",input[input_idx]);
					double temp_dist = 1000000.0;
					
					int face_id = 0;
					int point_id = 0;
					
					// If has negative i boundary
					if (isKthBitSet(int(input[input_idx]), 1))
					{
					//	printf("Has negative i boundary. \n");
						double dist_0 = calcDist(double(i), \
									double(j), \
									double(row_id + 0.5), \
									double(col_id - 0.5));
						//printf("%f\n", dist_0);

						double dist_1 = calcDist(double(i), \
									double(j), \
									double(row_id), \
									double(col_id - 0.5));
						
						//printf("%f\n", dist_1);
						double dist_2 = calcDist(double(i), \
									double(j), \
									double(row_id - 0.5), \
									double(col_id - 0.5));
			
						//printf("%f\n", dist_2);
						if (dist_0 < temp_dist)
						{
							temp_dist = dist_0;
							face_id = 0;
							point_id = 0;
						}
						if (dist_1 < temp_dist)
						{
							temp_dist = dist_1;
							face_id = 0;
							point_id = 1;
						}
						
						if (dist_2 < temp_dist)
						{
							temp_dist = dist_2;
							face_id = 0;
							point_id = 2;
						}
					}

					// If has positive i boundary
					if (isKthBitSet(int(input[input_idx]), 4))
					{
					//	printf("Check: %d\n", input[input_idx] & 0x08);
					//	printf("Has positive i boundary. \n");
						double dist_0 = calcDist(double(i), \
									double(j), \
									double(row_id - 0.5), \
									double(col_id + 0.5));
						
						double dist_1 = calcDist(double(i), \
									double(j), \
									double(row_id), \
									double(col_id + 0.5));
						
						double dist_2 = calcDist(double(i), \
									double(j), \
									double(row_id + 0.5), \
									double(col_id + 0.5));
			
						if (dist_0 < temp_dist)
						{
							temp_dist = dist_0;
							face_id = 2;
							point_id = 0;

						}
						if (dist_1 < temp_dist)
						{
							temp_dist = dist_1;
							face_id = 2;
							point_id = 1;
						}
						
						if (dist_2 < temp_dist)
						{
							temp_dist = dist_2;
							face_id = 2;
							point_id = 2;
						}
					}
					
					// If has negative j boundary
					if (isKthBitSet(int(input[input_idx]), 2))
					{
					//	printf("Has negative j boundary. \n");
						double dist_0 = calcDist(double(i), \
									double(j), \
									double(row_id - 0.5), \
									double(col_id + 0.5));
					//	printf("%f\n", dist_0);		
						double dist_1 = calcDist(double(i), \
									double(j), \
									double(row_id - 0.5), \
									double(col_id));
						
					//	printf("%f\n", dist_1);		
						double dist_2 = calcDist(double(i), \
									double(j), \
									double(row_id - 0.5), \
									double(col_id - 0.5));
			
					//	printf("%f\n", dist_2);		
						if (dist_0 < temp_dist)
						{
					//		printf("dist_0\n");
							temp_dist = dist_0;
							face_id = 1;
							point_id = 0;

						}
						if (dist_1 < temp_dist)
						{
					//		printf("dist_1\n");
							temp_dist = dist_1;
							face_id = 1;
							point_id = 1;

						}
						
						if (dist_2 < temp_dist)
						{
					//		printf("dist_2\n");
							temp_dist = dist_2;
							face_id = 1;
							point_id = 2;

						}
					}
					
					// If has positive j boundary
					if (isKthBitSet(int(input[input_idx]), 5))
					{
					//	printf("Has positive j boundary. \n");
						double dist_0 = calcDist(double(i), \
									double(j), \
									double(row_id + 0.5), \
									double(col_id - 0.5));
						
						double dist_1 = calcDist(double(i), \
									double(j), \
									double(row_id + 0.5), \
									double(col_id));
						
						double dist_2 = calcDist(double(i), \
									double(j), \
									double(row_id + 0.5), \
									double(col_id + 0.5));
			
						if (dist_0 < temp_dist)
						{
							temp_dist = dist_0;
							face_id = 3;
							point_id = 0;

						}
						if (dist_1 < temp_dist)
						{
							temp_dist = dist_1;
							face_id = 3;
							point_id = 1;
						}
						
						if (dist_2 < temp_dist)
						{
							temp_dist = dist_2;
							face_id = 3;
							point_id = 2;

						}
					}
					//printf("\n");
						
					if (face_id == 0)
					{
						printf("(NEG_I, %d)  ", point_id);
					}
					if (face_id == 1)
					{
						printf("(NEG_J, %d)  ", point_id);
					}
					if (face_id == 2)
					{
						printf("(POS_I, %d)  ", point_id);
					}
					if (face_id == 3)
					{
						printf("(POS_J, %d)  ", point_id);
					}
				}
			}

			printf("\n");
		}
	}
}

double calcDist(double i, double j, double target_i, double target_j)
{
	double result = (i - target_i) * (i - target_i) + \
			(j - target_j) * (j - target_j);

	return sqrt(result);	
}

int check(int *ref, int *output, int length)
{
	int i;
	for(i = 0; i < length; i++)
	{
		if(ref[i] != output[i])
		{
			return 0;
		}
	}

	return 1;
}
