#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#include "volume.h"
#include "exhaustiveFT.h"
#include "maurer.h"
#include "maurer_GPU.cuh"

/*
	Testing Functions are Defined here
*/
void initVolume(uchar *, int, int, int);
void printVolume(uchar *, int, int, int);
int check(int *, int *, int);
void printResult(int *vol, int height, int width, int depth);

/*
	The following functions should be further optimized before putting into
	Plastimatch.	
*/

bool isKthBitSet(int n, int k) 
{ 
    if (n & (1 << (k - 1))) 
        return 1; 
    else
        return 0; 
}

/*
	TODO - Spacing 
*/
double calcDist(double i, double j, double k, \
		double target_i, double target_j, double target_k)
{
	double result = (i - target_i) * (i - target_i) + \
			(j - target_j) * (j - target_j) + \
			(k - target_k) * (k - target_k);

	return sqrt(result);	
}

double distToFacePoint(int unchanged, // which dimension stays unchanged
			double i, double j, double k, 
			double cfv_i, double cfv_j, double cfv_k)
{
	double dist = DBL_MAX;

	// When processing NEG/POS_J direction
	if (unchanged == 0)
	{
		double temp_j = cfv_j - 0.5;

		int add_j;
		for (add_j = 0; add_j < 3; add_j++)
		{
			double temp_k = cfv_k - 0.5;

			int add_k;
			for (add_k = 0; add_k < 3; add_k++)
			{
				double temp_dist = calcDist(i, j, k,
							cfv_i, 
							temp_j + 0.5 * add_j, 
							temp_k + 0.5 * add_k);
				if(temp_dist < dist)
				{
					dist = temp_dist;
				}

			}
		}
	}

	// When processing NEG/POS_I direction
	if (unchanged == 1)
	{
		double temp_i = cfv_i - 0.5;

		int add_i;
		for (add_i = 0; add_i < 3; add_i++)
		{
			
			double temp_k = cfv_k - 0.5;

			int add_k;
			for (add_k = 0; add_k < 3; add_k++)
			{
				double temp_dist = calcDist(i, j, k,
							temp_i + 0.5 * add_i, 
							cfv_j, 
							temp_k + 0.5 * add_k);
				if(temp_dist < dist)
				{
					dist = temp_dist;
				}
			}
		}
	}

	// When processing NEG/POS_K direction	
	if (unchanged == 2)
	{
		double temp_i = cfv_i - 0.5;

		int add_i;
		for (add_i = 0; add_i < 3; add_i++)
		{
			
			double temp_j = cfv_j - 0.5;

			int add_j;
			for (add_j = 0; add_j < 3; add_j++)
			{
				double temp_dist = calcDist(i, j, k,
							temp_i + 0.5 * add_i, 
							temp_j + 0.5 * add_j, 
							cfv_k);
				if(temp_dist < dist)
				{
					dist = temp_dist;
				}
			}
		}
	}
	
	return dist;
}

double distToClosetFacePointOfFV(int i, int j, int k,
			int cfv_i, int cfv_j, int cfv_k,
		       	int cfv_val)
{
	double dist = DBL_MAX;
	
	// If the CFV has negative i face
	if (isKthBitSet(cfv_val, 1))
	{
		double temp_dist = distToFacePoint(1, 
					(double)i, (double)j, (double)k, 
					(double)cfv_i, (double)cfv_j-0.5, (double)cfv_k);
		
		if (temp_dist < dist)
		{
			dist = temp_dist;
		}
	}
	
	// If the CFV has negative j face
	if (isKthBitSet(cfv_val, 2))
	{
		double temp_dist = distToFacePoint(0, 
					(double)i, (double)j, (double)k, 
					(double)cfv_i-0.5, (double)cfv_j, (double)cfv_k);
		
		if (temp_dist < dist)
		{
			dist = temp_dist;
		}
	}
	
	// If the CFV has negative k face
	if (isKthBitSet(cfv_val, 3))
	{
		double temp_dist = distToFacePoint(2, 
					(double)i, (double)j, (double)k, 
					(double)cfv_i, (double)cfv_j, (double)cfv_k-0.5);

		if (temp_dist < dist)
		{
			dist = temp_dist;
		}
	}
	
	// If the CFV has positive i face
	if (isKthBitSet(cfv_val, 4))
	{
		double temp_dist = distToFacePoint(1, 
					(double)i, (double)j, (double)k, 
					(double)cfv_i, (double)cfv_j+0.5, (double)cfv_k);

		if (temp_dist < dist)
		{
			dist = temp_dist;
		}
	}
	
	// If the CFV has positive j face
	if (isKthBitSet(cfv_val, 5))
	{
		double temp_dist = distToFacePoint(0, 
					(double)i, (double)j, (double)k, 
					(double)cfv_i+0.5, (double)cfv_j, (double)cfv_k);

		if (temp_dist < dist)
		{
			dist = temp_dist;
		}
	}

	// If the CFV has positive k face
	if (isKthBitSet(cfv_val, 6))
	{
		double temp_dist = distToFacePoint(2, 
					(double)i, (double)j, (double)k, 
					(double)cfv_i, (double)cfv_j, (double)cfv_k+0.5);

		if (temp_dist < dist)
		{
			dist = temp_dist;
		}
	}

	return dist;
}



int main()
{
	// Initialize Input and Output Data
	/*	ref will always stay in host */
	int *ref = (int *)malloc(HEIGHT * WIDTH * DEPTH * sizeof(int));
	if (ref == NULL)
	{
		printf("malloc() failed! \n");
		return 0;
	}

	/* Input will be transfered to GPU */
	uchar *input;
	if (hipHostAlloc((void **)&input, HEIGHT * WIDTH * DEPTH * sizeof(uchar), \
						hipHostMallocDefault) != hipSuccess)
	{
		printf("hipHostAlloc() failed! \n");
		return 0;
	}

	/* Output will be transfered back to CPU */
	int *output_0;
	if (hipHostAlloc((void **)&output_0, HEIGHT * WIDTH * DEPTH * sizeof(int), \
						hipHostMallocDefault) != hipSuccess)
	{
		printf("hipHostAlloc() failed! \n");
		return 0;
	}

	int *output_1;
	if (hipHostAlloc((void **)&output_1, HEIGHT * WIDTH * DEPTH * sizeof(int), \
						hipHostMallocDefault) != hipSuccess)
	{
		printf("hipHostAlloc() failed! \n");
		return 0;
	}

	for (int i = 0; i < HEIGHT * WIDTH * DEPTH; i++)
	{
		output_0[i] = -1;
		output_1[i] = -1;
		ref[i] = -1;
	}

	// Generate Testing Data
	initVolume(input, HEIGHT, WIDTH, DEPTH);	

	// Print Volume
	printVolume(input, HEIGHT, WIDTH, DEPTH);

	// Generate Reference Result
	exhaustiveFT(input, HEIGHT, WIDTH, DEPTH, ref);

	

	// Compute using Maurer
	struct timeval stopCPU, startCPU;
	gettimeofday(&startCPU, NULL);
	maurerFT(input, HEIGHT, WIDTH, DEPTH, output_0);
	gettimeofday(&stopCPU, NULL);
	long seconds = stopCPU.tv_sec - startCPU.tv_sec;
	long useconds = stopCPU.tv_usec - startCPU.tv_usec;
	long mtime = seconds * 1000 + useconds / 1000.0;
	//printf("CPU Execution Time: %ld ms. \n", mtime);

	// Check the closet boundary point
		

	printResult(output_0, HEIGHT, WIDTH, DEPTH);

	/*
		GPU Solutions
	*/
	/* CUDA Performance */
	/*
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// Allocate device memory for input data
	uchar *dev_vol;
	if (hipMalloc((void **)&dev_vol, HEIGHT * WIDTH * DEPTH * sizeof(uchar)\
		) != hipSuccess)
	{
		printf("hipMalloc() failed.\n");
		return 0;
	}
	
	// Allocate device memory for ping-pong I/O
	int *dev_ping_pong_buf[2];

	for (int i = 0; i < 2; i++)
	{
		if (hipMalloc((void **)&(dev_ping_pong_buf[i]), HEIGHT * WIDTH * DEPTH * sizeof(int)\
			) != hipSuccess)
		{
			printf("hipMalloc() failed! \n");
			return 0;
		}	
	}

	// Bind texture memory - Remember to Unbind!
	// if (hipBindTexture( NULL, ping_pong_tex_0, dev_ping_pong_buf[0], HEIGHT * WIDTH * DEPTH * sizeof(int)) \
		!= hipSuccess)
	// {
	//	printf("hipBindTexture() failed! \n");
	//	return 0;
	// }

	// Copy input data to GPU
	hipMemcpy(dev_vol, input, HEIGHT * WIDTH * DEPTH * sizeof(uchar), \
		hipMemcpyHostToDevice);

	// Computation
	dim3 dimBlock_D1_V1(256, 2, 2);
	dim3 dimGrid_D1_V1(1, 128, 32);

	dim3 dimBlock_D2_V1(2, 256, 2);
	dim3 dimGrid_D2_V1(128, 1, 32);

	//dim3 dimBlock_D3(4, 4, 64);
	//dim3 dimGrid_D3(64, 64, 1);

	dim3 dimBlock_D1_V2(32, 8, 4);
	dim3 dimGrid_D1_V2(1, 4, 2);
	//dim3 dimGrid_D1_V2(1, 32, 16);

	dim3 dimBlock_D2_V2(8, 32, 4);
	dim3 dimGrid_D2_V2(4, 1, 2);
	//dim3 dimGrid_D2_V2(32, 1, 16);

	//dim3 dimBlock_D3_V2(8, 4, 32);
	//dim3 dimGrid_D3_V2(32, 64, 1);
	dim3 dimBlock_D3_V2(8, 4, 8);
	dim3 dimGrid_D3_V2(4, 8, 1);


	hipEventRecord(start);
	//initVol_GPU<<<360, 256>>>(dev_vol, HEIGHT, WIDTH, DEPTH, dev_output);
	//raster_scan_GPU_v1<<<dimGrid_D1_V1, dimBlock_D1_V1>>>(dev_vol, 1, WIDTH, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[0]);
	raster_scan_GPU_v2<<<dimGrid_D1_V2, dimBlock_D1_V2>>>(dev_vol, 0, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[1]);
	raster_scan_GPU_v2<<<dimGrid_D2_V2, dimBlock_D2_V2>>>(dev_ping_pong_buf[1], 1, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[0]);
	raster_scan_GPU_v2<<<dimGrid_D3_V2, dimBlock_D3_V2>>>(dev_ping_pong_buf[0], 2, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[1]);

	//raster_scan_GPU_v2<<<dimGrid_D3_V2, dimBlock_D3_V2>>>(dev_ping_pong_buf[0], 2, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[1]);
	//raster_scan_GPU_v1<<<dimGrid_D2_V1, dimBlock_D2_V1>>>(dev_ping_pong_buf[0], 2, HEIGHT, HEIGHT, WIDTH, DEPTH, \
												dev_ping_pong_buf[1]);
	// Different buffer is needed since the original data may get overwritten
	//raster_scan_GPU_v2<<<dimGrid_D2_V2, dimBlock_D2_V2>>>(dev_ping_pong_buf[0], 1, HEIGHT, WIDTH, DEPTH, \
															dev_ping_pong_buf[1]);

	// maurer_GPU<<<dimGrid_D3, dimBlock_D3>>>(dev_vol, 3, DEPTH, HEIGHT, WIDTH, DEPTH, dev_output);
	
	// Send result back to host
	//hipMemcpy(output_0, dev_ping_pong_buf[0], HEIGHT * WIDTH * DEPTH * sizeof(int), \
		hipMemcpyDeviceToHost);
	hipMemcpy(output_1, dev_ping_pong_buf[1], HEIGHT * WIDTH * DEPTH * sizeof(int), \
		hipMemcpyDeviceToHost);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Execution Time on GPU: %f ms\n", milliseconds);

	// Free device resource
	// hipUnbindTexture(ping_pong_tex_0);

	hipFree(dev_vol);
	for (int i = 0; i < 2; i++)
	{
		hipFree(dev_ping_pong_buf[i]);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	*/

	// Check result
	if (check(ref, output_0, HEIGHT * WIDTH * DEPTH) == 0)
	{
		printf("Error! \n");
	}
	else
	{
		printf("Successful! \n");
	}

	/*
	if (check(ref, output_1, HEIGHT * WIDTH * DEPTH) == 0)
	{
		printf("GPU V2 Error! \n");
	}
	else
	{
		printf("GPU V2 Successful! \n");
	}
	*/
	
	// De-allocate Memory
	hipHostFree(input);
	hipHostFree(output_0);
	hipHostFree(output_1);
	free(ref);	
}

void initVolume(uchar *vol, int height, int width, int depth)
{
	for (int i = 0; i < HEIGHT * WIDTH * DEPTH; i++)
	{
		 vol[i] = 0;
	}

	srand(time(NULL));

	// Distance between slices
	int slice_stride = height * width;

	/*
	for (int ite = 0; ite < 5; ite++)
	{
		int r_row = rand() % height;
		int r_col = rand() % width;
		int r_dep = rand() % depth;

		vol[r_dep * slice_stride + r_row * width + r_col] = 1;
	}
	*/
	
	int r_row = 3;
	int r_col = 3;
	int r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x03;

	r_row = 3;
	r_col = 4;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x02;

	r_row = 3;
	r_col = 5;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x0A;

	r_row = 4;
	r_col = 3;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x01;

	r_row = 4;
	r_col = 6;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x0A;

	r_row = 5;
	r_col = 3;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x11;

	r_row = 5;
	r_col = 6;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x08;

	r_row = 6;
	r_col = 4;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x11;
	
	r_row = 6;
	r_col = 5;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x10;

	r_row = 6;
	r_col = 6;
	r_dep = 0;
	vol[r_dep * slice_stride + r_row * width + r_col] = 0x18;
}

void printVolume(uchar *vol, int height, int width, int depth)
{
	// Distance between slices
	int slice_stride = height * width;

	int i, j, k;

	for (k = 0; k < depth; k++)
	{
		printf("Image Slice: %d\n", k);

		for (i = 0; i < height; i++)
		{
			for (j = 0; j < width; j++)
			{
				printf("0x%02x ", vol[k * slice_stride + i * width + j]);
			}
			printf("\n");
		}
		printf("\n");
	}
}

void printResult(int *vol, int height, int width, int depth)
{
	int slice_stride = height * width;

	int i, j, k;

	for (k = 0; k < depth; k++)
	{
		printf("Image Slice: %d\n", k);

		for (i = 0; i < height; i++)
		{
			for (j = 0; j < width; j++)
			{
				int row_id = vol[k * slice_stride + i * width + j] / width;
				int col_id = vol[k * slice_stride + i * width + j] % width;
				
				if (row_id == i && col_id == j)
				{
					printf("****** ");
				}
				else
				{
					printf("(%d, %d) ", row_id, col_id);
				}
			}
			printf("\n");
		}
		printf("\n");	
	}
}

int check(int *ref, int *output, int length)
{
	int i;
	for(i = 0; i < length; i++)
	{
		if(ref[i] != output[i])
		{
			return 0;
		}
	}

	return 1;
}
